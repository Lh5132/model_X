#define CU
#include "cuda_functions.cuh"
#include "node.h"
#include "hip/hip_runtime.h"
#include ""

namespace model_X
{
	void node_to_cuda(node* input)
	{
		DTYPE* temp_data = 0;
		size_t t_size = input->total_size*DBYTES;
		hipMalloc((void**)&input->cuda_data, sizeof(node));
		hipMalloc((void**)&temp_data, t_size);
		hipMemcpy(input->cuda_data, input, sizeof(node), hipMemcpyHostToDevice);
		hipMemcpy(temp_data, input->data, t_size, hipMemcpyHostToDevice);
		hipMemcpy(&input->cuda_data->data, &temp_data, sizeof(void*), hipMemcpyHostToDevice);
	}
	void node_to_cpu(node* input)
	{
		DTYPE* temp_data = 0;
		hipMemcpy(&temp_data, &input->cuda_data->data, sizeof(void*), hipMemcpyDeviceToHost);
		hipMemcpy(input->data, temp_data, input->total_size*DBYTES, hipMemcpyDeviceToHost);
		hipFree(input->cuda_data);
	}
}